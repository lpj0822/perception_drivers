#include "hip/hip_runtime.h"
/*!
* \brief This file defines image gpu class for objects parsing and transferring
* \attention Copyright © Holomatic Technology (Beijing) Co.Ltd
* \attention Please refer to COPYRIGHT.txt for complete terms of copyright notice. */

#include <holo/sensors/camera/image_gpu.h>

inline __device__ __host__ float clamp(float f, float a, float b)
{
    return fmaxf(a, fminf(f, b));
}


/* From RGB to YUV

   Y = 0.299R + 0.587G + 0.114B
   U = 0.492 (B-Y)
   V = 0.877 (R-Y)

   It can also be represented as:

   Y =  0.299R + 0.587G + 0.114B
   U = -0.147R - 0.289G + 0.436B
   V =  0.615R - 0.515G - 0.100B

   From YUV to RGB

   R = Y + 1.140V
   G = Y - 0.395U - 0.581V
   B = Y + 2.032U
 */

struct __align__(2) uchar6
{
   uchar3 a0, a1;
};
static __host__ __device__ __forceinline__ uchar6 make_uchar6(uchar3 a0, uchar3 a1)
{
   uchar6 val = {a0, a1};
   return val;
}

//__global__ void yuyvToRgb( uchar4* src, int srcAlignedWidth, uchar6* dst, int dstAlignedWidth, int width, int height )
__global__ void yuyvToRgb(uint8_t* yuyv, uint8_t* rgb, int srcAlignedWidth, int dstAlignedWidth, int width, int height )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= srcAlignedWidth || y >= height )
		return;

	if(y* srcAlignedWidth +x >= 1920/1080*2/4)
	{
	    //printf("there is a boundary %d\n", y* srcAlignedWidth +x);
	    //return;
	}
	const uchar4 macroPx = ((uchar4*)yuyv)[y * 1920/2 + x];// = src[y * srcAlignedWidth + x];
	

	// Y0 is the brightness of pixel 0, Y1 the brightness of pixel 1.
	// U0 and V0 is the color of both pixels.
	// UYVY [ U0 | Y0 | V0 | Y1 ] 
	// YUYV [ Y0 | U0 | Y1 | V0 ]
	const float y0 = macroPx.x;
	const float y1 = macroPx.z; 
	const float u = (macroPx.y) - 128.0f;
	const float v = (macroPx.w) - 128.0f;

	const float3 px0 = make_float3( y0 + 1.4065f * v,
				        y0 - 0.3455f * u - 0.7169f * v,
					y0 + 1.7790f * u);

	const float3 px1 = make_float3( y1 + 1.4065f * v,
					y1 - 0.3455f * u - 0.7169f * v,
					y1 + 1.7790f * u);

	((uchar6*)rgb)[y * 1920/2 + x] = make_uchar6( make_uchar3(clamp(px0.x, 0.0f, 255.0f), 
							       clamp(px0.y, 0.0f, 255.0f),
							       clamp(px0.z, 0.0f, 255.0f)),
                                                    make_uchar3(clamp(px1.x, 0.0f, 255.0f), 
							       clamp(px1.y, 0.0f, 255.0f),
							       clamp(px1.z, 0.0f, 255.0f)));
     
} 


void CudaYUYVToRGB(const int32_t& cols, const int32_t& rows, uint8_t* cuda_yuyv, uint8_t* cuda_rgb, uint8_t* yuyv, uint8_t* rgbb_image)
{
	hipError_t cuda_status;

	cuda_status = hipSetDevice(0);
	if(cuda_status != hipSuccess)
	{
	    printf( "Device does not support cuda\n");
	    return;// cuda_status;
	}

	cuda_status = hipMemcpy(cuda_yuyv, yuyv, cols * rows * 2, hipMemcpyHostToDevice);
	if(cuda_status != hipSuccess)
	{
	    printf( "Could not copy data to gpu\n");
	    return;// cuda_status;
	}

	dim3 block(8,8);
	dim3 grid(cols/2/8, rows/8);
        const int srcAlignedWidth = cols*rows*2 / sizeof(uchar4);	// normally would be uchar2, but we're doubling up pixels
	const int dstAlignedWidth = cols*rows*3 / sizeof(uchar6);	// normally would be uchar4 ^^^

	//printf("yuyvToRgba %zu %zu %i %i %i %i %i\n", width, height, (int)formatUYVY, srcAlignedWidth, dstAlignedWidth, grid.x, grid.y);

	//yuyvToRgb<<<grid, block>>>((uchar4*)cuda_yuyv, srcAlignedWidth, (uchar6*)cuda_rgb, dstAlignedWidth, cols, rows);
	yuyvToRgb<<<grid, block>>>(cuda_yuyv, cuda_rgb, srcAlignedWidth, dstAlignedWidth, cols, rows);

        cuda_status = hipGetLastError();
        
        if(cuda_status != hipSuccess)
	{
	    printf( "Yuyv to rgb error\n");
	    return;// cuda_status;
	}

        cuda_status = hipDeviceSynchronize();
        if(cuda_status != hipSuccess)
	{
	    printf("sychronization error\n");
	    return;// cuda_status;
	}

        cuda_status = hipMemcpy(rgbb_image, cuda_rgb, cols*rows*3, hipMemcpyDeviceToHost);
        if(cuda_status != hipSuccess)
	{
	    printf("Could not copy data from gpu\n");
	    return;// cuda_status;
	}

        return;// cuda_status;
}